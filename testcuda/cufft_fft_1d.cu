
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#ifndef pi
#define pi 4.0f*atanf(1.0f)
#endif
#ifndef threads_num
#define threads_num 256
#endif

static __global__ void DataTypeConvertFloatToComplex_1d(float *d_in, hipfftComplex *d_out, int n);
static __global__ void DataGetBackFft_1d(hipfftComplex *d_in, float *d_out, float dx, int n);
static __global__ void PhaseShiftForwFft_1d(hipfftComplex *d_in, hipfftComplex *d_out, int n);
static __global__ void PhaseShiftBackFft_1d(hipfftComplex *d_in, hipfftComplex *d_out, int n);

void cufft_fftf_1d(float *d_in, float *d_out, float dx, int n)
{
	/* Data conversion from float to Complex */
	hipfftComplex *d_in_data;
	hipMalloc((void **)&d_in_data, n*sizeof(hipfftComplex));
	DataTypeConvertFloatToComplex_1d<<<(n+threads_num-1)/threads_num, threads_num>>>(d_in, d_in_data, n);

	/* Create a 1D forward/inverse FFT plan */
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);

	/* Use the CUFFT plan to transform the array */
	hipfftComplex *d_tmp_data;
	hipMalloc((void **)&d_tmp_data, n*sizeof(hipfftComplex));
	hipfftExecC2C(plan, d_in_data, d_tmp_data, HIPFFT_FORWARD);

	/* 1/2 phase shift for staggered grid */
	hipfftComplex *d_tmp_data_shift;
	hipMalloc((void **)&d_tmp_data_shift, n*sizeof(hipfftComplex));
	PhaseShiftForwFft_1d<<<(n+threads_num-1)/threads_num, threads_num>>>(d_tmp_data, d_tmp_data_shift, n);

	/* Inverse fourier transform */
	hipfftComplex *d_out_data;
	hipMalloc((void **)&d_out_data, n*sizeof(hipfftComplex));
	hipfftExecC2C(plan, d_tmp_data_shift, d_out_data, HIPFFT_BACKWARD);

	/* Data normalization*/
	DataGetBackFft_1d<<<(n+threads_num-1)/threads_num, threads_num>>>(d_out_data, d_out, dx, n);

	hipfftDestroy(plan);
	hipFree(d_in_data);
	hipFree(d_tmp_data);
	hipFree(d_tmp_data_shift);
	hipFree(d_out_data);
}

void cufft_fftb_1d(float *d_in, float *d_out, float dx, int n)
{
	/* Data conversion from float to Complex */
	hipfftComplex *d_in_data;
	hipMalloc((void **)&d_in_data, n*sizeof(hipfftComplex));
	DataTypeConvertFloatToComplex_1d<<<(n+threads_num-1)/threads_num, threads_num>>>(d_in, d_in_data, n);

	/* Create a 1D forward/inverse FFT plan */
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);

	/* Use the CUFFT plan to transform the array */
	hipfftComplex *d_tmp_data;
	hipMalloc((void **)&d_tmp_data, n*sizeof(hipfftComplex));
	hipfftExecC2C(plan, d_in_data, d_tmp_data, HIPFFT_FORWARD);

	/* -1/2 phase shift for staggered grid */
	hipfftComplex *d_tmp_data_shift;
	hipMalloc((void **)&d_tmp_data_shift, n*sizeof(hipfftComplex));
	PhaseShiftBackFft_1d<<<(n+threads_num-1)/threads_num, threads_num>>>(d_tmp_data, d_tmp_data_shift, n);

	/* Inverse fourier transform */
	hipfftComplex *d_out_data;
	hipMalloc((void **)&d_out_data, n*sizeof(hipfftComplex));
	hipfftExecC2C(plan, d_tmp_data_shift, d_out_data, HIPFFT_BACKWARD);

	/* Data normalization*/
	DataGetBackFft_1d<<<(n+threads_num-1)/threads_num, threads_num>>>(d_out_data, d_out, dx, n);

	hipfftDestroy(plan);
	hipFree(d_in_data);
	hipFree(d_tmp_data);
	hipFree(d_tmp_data_shift);
	hipFree(d_out_data);
}

static __global__ void DataTypeConvertFloatToComplex_1d(float *d_in, hipfftComplex *d_out, int n)
{
	//int tid = threadIdx.x;
	int ix = blockIdx.x * blockDim.x + threadIdx.x;

	//for (int ix=tid; ix<n; ix+=threads_num)
	if (ix < n)
	{
		d_out[ix].x = d_in[ix];
		d_out[ix].y = 0.0f;
	}
}

static __global__ void DataGetBackFft_1d(hipfftComplex *d_in, float *d_out, float dx, int n)
{
	//int tid = threadIdx.x;
	int ix = blockIdx.x * blockDim.x + threadIdx.x;

	//for (int ix=tid; ix<n; ix+=threads_num)
	if (ix < n)
		d_out[ix] = d_in[ix].x/((float)n*dx);
}

static __global__ void PhaseShiftForwFft_1d(hipfftComplex *d_in, hipfftComplex *d_out, int n)
{
	//int tid = threadIdx.x;
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	float d_k;

	//for (int ix=tid; ix<n; ix+=threads_num)
	if (ix < n)
	{
		if (ix<n/2)
			d_k = (float)ix*pi/(float)(n/2);
		else
			d_k = -pi+(float)(ix-n/2)*pi/(float)(n/2);
		
		d_out[ix].y = d_k*(d_in[ix].x*cosf(d_k/2.0f)+d_in[ix].y*sinf(d_k/2.0f));
		d_out[ix].x = d_k*(-d_in[ix].x*sinf(d_k/2.0f)+d_in[ix].y*cosf(d_k/2.0f));
	}
}

static __global__ void PhaseShiftBackFft_1d(hipfftComplex *d_in, hipfftComplex *d_out, int n)
{
	//int tid = threadIdx.x;
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	float d_k;

	//for (int ix=tid; ix<n; ix+=threads_num)
	if (ix < n)
	{
		if (ix<n/2)
			d_k = (float)ix*pi/(float)(n/2);
		else
			d_k = -pi+(float)(ix-n/2)*pi/(float)(n/2);
		
		d_out[ix].y = d_k*(d_in[ix].x*cosf(d_k/2.0f)-d_in[ix].y*sinf(d_k/2.0f));
		d_out[ix].x = d_k*(d_in[ix].x*sinf(d_k/2.0f)+d_in[ix].y*cosf(d_k/2.0f));
	}
}

void data_fft_derv(float *h_in, float *h_out, float dx, int n)
{
	float *d_in;
	hipMalloc((void **)&d_in, n*sizeof(float));
	hipMemcpy(d_in, h_in, n*sizeof(float), hipMemcpyHostToDevice);
	
	float *d_out;
	hipMalloc((void **)&d_out, n*sizeof(float));
	
	cufft_fftb_1d(d_in, d_out, dx, n);

	hipMemcpy(h_out, d_out, n*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
}
